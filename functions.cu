#include "hip/hip_runtime.h"
#include "fashion.h"

// Input functions: no optimizations
int reverse_int32(int i) {
  unsigned char byte1, byte2, byte3, byte4;
  byte1 = i & MAXBYTE;
  byte2 = (i >> 8) & MAXBYTE;
  byte3 = (i >> 16) & MAXBYTE;
  byte4 = (i >> 24) & MAXBYTE;
  return ((int)byte1 << 24) + ((int)byte2 << 16) + ((int)byte3 << 8) +
         (int)byte4;
}
// Input functions: no optimizations
void read_data(const char *datapath, host_vector<host_vector<float>> &data) {
  ifstream infile(datapath, ios::binary);
  if (!infile.is_open()) {
    printf("FAILED TO OPEN FILE: %s\n", datapath);
    return;
  }
  cout << "== Input test image file: " << datapath << endl;
  // Read the header information
  int magic_number = 0;
  int number_of_images = 0;
  int n_rows = 0;
  int n_cols = 0;
  infile.read((char *)&magic_number, sizeof(magic_number));
  magic_number = reverse_int32(magic_number);
  cout << "magic number: " << magic_number << endl;
  infile.read((char *)&number_of_images, sizeof(number_of_images));
  number_of_images = reverse_int32(number_of_images);
  cout << "number of images: " << number_of_images << endl;
  infile.read((char *)&n_rows, sizeof(n_rows));
  n_rows = reverse_int32(n_rows);
  infile.read((char *)&n_cols, sizeof(n_cols));
  n_cols = reverse_int32(n_cols);
  n_rows += 4;
  n_cols += 4;
  cout << "size of row = " << n_rows << ", size of cols = " << n_cols << endl;
  // Read actual data (uint8 -> float)
  for (int i = 0; i < number_of_images / MINIBATCH; ++i) {
    for (int n = 0; n < MINIBATCH; n++) {
      for (int r = 2; r < n_rows - 2; ++r) {
        for (int c = 2; c < n_cols - 2; ++c) {
          unsigned char temp = 0;
          infile.read((char *)&temp, sizeof(temp));
          data[i][(n_cols * n_rows * n) + (n_rows * r) + c] =
              (float)temp * (float)MNIST_SCALE_FACTOR;
        }
      }
    }
  }
  infile.close();
  cout << "Done. [data: " << datapath << "] [count: " << number_of_images << "]"
       << endl;
}
// Input functions: no optimizations
void read_label(const char *labelPath, host_vector<int> &labels) {
  int number_of_labels = 0;
  ifstream infile(labelPath, ios::binary);
  if (!infile.is_open()) {
    printf("FAILED TO OPEN FILE: %s\n", labelPath);
    return;
  }
  cout << "== Input test label file: " << labelPath << endl;
  int magic_number = 0;
  // read the label information
  infile.read((char *)&magic_number, sizeof(magic_number));
  magic_number = reverse_int32(magic_number);
  cout << "magic number: " << magic_number << endl;
  infile.read((char *)&number_of_labels, sizeof(number_of_labels));
  number_of_labels = reverse_int32(number_of_labels);
  cout << "number of labels: " << number_of_labels << endl;
  for (int i = 0; i < number_of_labels; ++i) {
    unsigned char temp = 0;
    infile.read((char *)&temp, sizeof(temp));
    labels[i] = (int)temp;
  }
  infile.close();
  cout << "Done. [data: " << labelPath << "] [count: " << number_of_labels
       << "] " << endl;
}

// ReLU kernel in forward propagation
__global__ void relu_h(float *X, float *Y, int size_in) {
  int t = blockIdx.x * blockDim.x + threadIdx.x;
  if (t < size_in) {
    Y[t] = 0.0;
    if (X[t] >= 0)
      Y[t] = X[t];
  }
}

// ReLU in forward propagation
void forward_relu(device_vector<float> &input, device_vector<float> &output) {
  int size_in = input.size();
  float *input_pointer = thrust::raw_pointer_cast(input.data());
  float *output_pointer = thrust::raw_pointer_cast(output.data());
  int block_size = ceil((double)size_in / 1024);
  relu_h<<<block_size, 1024>>>(input_pointer, output_pointer, size_in);
}

// ReLU kernel in backward propagation
__global__ void backward_relu_h(float *X, float *Y, int size_in) {
  int t = blockIdx.x * blockDim.x + threadIdx.x;
  if (t < size_in) {
    X[t] = 0.0;
    if (X[t] >= 0)
      X[t] = Y[t];
  }
}

// ReLU in backward propagation
void backward_relu(device_vector<float> &input, device_vector<float> &output) {
  int size_in = input.size();
  float *input_pointer = thrust::raw_pointer_cast(input.data());
  float *output_pointer = thrust::raw_pointer_cast(output.data());
  int block_size = ceil((double)size_in / 1024);
  backward_relu_h<<<block_size, 1024>>>(input_pointer, output_pointer, size_in);
}

// Matrix reduction kernel
__global__ void reduce_to_first_index_h(float *X, int height, int width) {
  int t = blockIdx.x * blockDim.x + threadIdx.x;
  float tmp = 0;
  if (t < width) {
    for (int i = 0; i < height; i++) {
      tmp += X[i * width + t];
    }
    X[t] = tmp;
  }
}

// Matrix reduction: pointer version
void reduce_to_first_index(float *input_pointer, int height, int width) {
  int block_size = ceil((float)width / 1024);
  reduce_to_first_index_h<<<block_size, 1024>>>(input_pointer, height, width);
}

// Matrix reduction: device vector version
void reduce_to_first_index(device_vector<float> &input, int height, int width) {
  float *input_pointer = thrust::raw_pointer_cast(input.data());
  int block_size = ceil((float)width / 1024);
  reduce_to_first_index_h<<<block_size, 1024>>>(input_pointer, height, width);
}

// Add bias to all values kernel
__global__ void forward_bias(float *X, float *b, int N, int ch_in, int h_in,
                             int w_in) {
  int n = blockIdx.x;
  int ch = blockIdx.y;
  int h = threadIdx.x;
  int w = threadIdx.y;
  X[n * ch_in * h_in * w_in + ch * h_in * w_in + h * w_in + w] += b[ch];
}

// Add bias
void forward_bias_per_channel(device_vector<float> &input,
                              device_vector<float> &bias, int N, int ch_in,
                              int h_in, int w_in) {
  dim3 blockDim(N, ch_in);
  dim3 threadDim(h_in, w_in);
  float *input_pointer = thrust::raw_pointer_cast(input.data());
  float *bias_pointer = thrust::raw_pointer_cast(bias.data());
  forward_bias<<<blockDim, threadDim>>>(input_pointer, bias_pointer, N, ch_in,
                                        h_in, w_in);
}

// General matrix multiplication
__global__ void gemm_h(float *Md, float *Nd, float *Pd, int M_height_in,
                       int M_width_N_height_in, int N_width_in, int height_out,
                       int width_out) {
  __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int row = by * TILE_WIDTH + ty;
  int col = bx * TILE_WIDTH + tx;
  float Pvalue = 0;
  for (int m = 0; m < ceilf((float)M_width_N_height_in / TILE_WIDTH); ++m) {
    int mtx = m * TILE_WIDTH + tx;
    int mty = m * TILE_WIDTH + ty;
    if (row < M_height_in && mtx < M_width_N_height_in)
      Mds[ty][tx] = Md[row * M_width_N_height_in + mtx];
    else
      Mds[ty][tx] = 0;
    if (mty < M_width_N_height_in && col < N_width_in)
      Nds[ty][tx] = Nd[mty * N_width_in + col];
    else
      Nds[ty][tx] = 0;
    __syncthreads();
    for (int k = 0; k < TILE_WIDTH; ++k) {
      Pvalue += Mds[ty][k] * Nds[k][tx];
    }
    __syncthreads();
  }
  if (row < height_out && col < width_out)
    Pd[row * width_out + col] = Pvalue;
}
// General matrix multiplication with bias
__global__ void gemm_with_bias_h(float *Md, float *Nd, float *Pd, float *B,
                                 int M_height_in, int M_width_N_height_in,
                                 int N_width_in, int height_out,
                                 int width_out) {
  __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int row = by * TILE_WIDTH + ty;
  int col = bx * TILE_WIDTH + tx;
  float Pvalue = 0;
  for (int m = 0; m < ceilf((float)M_width_N_height_in / TILE_WIDTH); ++m) {
    int mtx = m * TILE_WIDTH + tx;
    int mty = m * TILE_WIDTH + ty;
    if (row < M_height_in && mtx < M_width_N_height_in)
      Mds[ty][tx] = Md[row * M_width_N_height_in + mtx];
    else
      Mds[ty][tx] = 0;
    if (mty < M_width_N_height_in && col < N_width_in)
      Nds[ty][tx] = Nd[mty * N_width_in + col];
    else
      Nds[ty][tx] = 0;
    __syncthreads();
    for (int k = 0; k < TILE_WIDTH; ++k) {
      Pvalue += Mds[ty][k] * Nds[k][tx];
    }
    __syncthreads();
  }
  if (row < height_out && col < width_out)
    Pd[row * width_out + col] = Pvalue + B[col];
}

// Matrix multiplication kernel
__global__ void transposeMatrix_h(float *odata, float *idata, int height,
                                  int width) {
  __shared__ float block[TILE_WIDTH][TILE_WIDTH + 1];
  unsigned int xIndex = blockIdx.x * TILE_WIDTH + threadIdx.x;
  unsigned int yIndex = blockIdx.y * TILE_WIDTH + threadIdx.y;
  if ((xIndex < width) && (yIndex < height)) {
    unsigned int index_in = yIndex * width + xIndex;
    block[threadIdx.y][threadIdx.x] = idata[index_in];
  }
  __syncthreads();
  xIndex = blockIdx.y * TILE_WIDTH + threadIdx.x;
  yIndex = blockIdx.x * TILE_WIDTH + threadIdx.y;
  if ((xIndex < height) && (yIndex < width)) {
    unsigned int index_out = yIndex * height + xIndex;
    odata[index_out] = block[threadIdx.x][threadIdx.y];
  }
}

// Matrix multiplication: device vector version
void transposeMatrix(device_vector<float> &outputT, device_vector<float> &input,
                     int input_height, int input_width) {
  dim3 threadsPerBlock(TILE_WIDTH, TILE_WIDTH);
  dim3 numBlocks_transpose_X(ceil((float)input_width / TILE_WIDTH),
                             ceil((float)input_height / TILE_WIDTH));
  float *XT_pointer = thrust::raw_pointer_cast(outputT.data());
  float *X_pointer = thrust::raw_pointer_cast(input.data());
  transposeMatrix_h<<<numBlocks_transpose_X, threadsPerBlock>>>(
      XT_pointer, X_pointer, input_height, input_width);
}

// Matrix multiplication: pointer version
void transposeMatrix(float *XT_pointer, float *X_pointer, int input_height,
                     int input_width) {
  dim3 threadsPerBlock(TILE_WIDTH, TILE_WIDTH);
  dim3 numBlocks_transpose_X(ceil((float)input_width / TILE_WIDTH),
                             ceil((float)input_height / TILE_WIDTH));
  transposeMatrix_h<<<numBlocks_transpose_X, threadsPerBlock>>>(
      XT_pointer, X_pointer, input_height, input_width);
}

// Gradient descent
__global__ void grad_descent(float *odata, const float *idata, int size) {
  int t = blockIdx.x * blockDim.x + threadIdx.x;
  if (t < size) {
    odata[t] -= LEARNIG_RATE * idata[t];
  }
}
// Backward propagation
void backward_bias_per_channel(device_vector<float> &input,
                               device_vector<float> &bias, int N, int h_in,
                               int w_total_in, int w_ch,
                               int w_width_mul_w_height) {
  device_vector<float> input_tmp, input_per_minibatch_T;
  input_tmp.resize(h_in * w_total_in, 0);
  input_per_minibatch_T.resize(h_in * w_total_in, 0);
  input_tmp = input;

  float *input_pointer = thrust::raw_pointer_cast(input_tmp.data());
  float *input_per_minibatch_T_pointer =
      thrust::raw_pointer_cast(input_per_minibatch_T.data());
  float *bias_pointer = thrust::raw_pointer_cast(bias.data());

  for (int i = 0; i < N; i++) {
    transposeMatrix(input_per_minibatch_T_pointer, input_pointer, w_ch,
                    w_width_mul_w_height);
    reduce_to_first_index(input_per_minibatch_T_pointer, w_width_mul_w_height,
                          w_ch);
    input_pointer += w_total_in;
    input_per_minibatch_T_pointer += w_total_in;
  }

  input_per_minibatch_T_pointer =
      thrust::raw_pointer_cast(input_per_minibatch_T.data());
  reduce_to_first_index(input_per_minibatch_T_pointer, h_in, w_total_in);

  int blockDim_b = ceil((float)w_ch / 1024);
  thrust::transform(input_per_minibatch_T.begin(), input_per_minibatch_T.end(),
                    input_per_minibatch_T.begin(), div_h());
  grad_descent<<<blockDim_b, 1024>>>(bias_pointer,
                                     input_per_minibatch_T_pointer, w_ch);
}
