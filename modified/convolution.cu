#include "hip/hip_runtime.h"
#include "fashion.h"

// Convolution layer initialization
void Convolution::init(int minibatch, int input_image_h, int input_image_w,
                       int W_w_h, int W_ch) {
  // Define random generator for initializing weights
  std::default_random_engine generator;
  std::normal_distribution<float> distribution(0, 0.1);
  // Initialize the member variables
  this->W_width_height = W_w_h;
  this->W_channel = W_ch;
  this->X_width = input_image_h * input_image_w;
  this->X_height = minibatch;
  this->input_image_width = input_image_w;
  this->input_image_height = input_image_h;
  this->minibatch = minibatch;
  this->Outputimage_width = (input_image_width - W_width_height + 1);
  this->Outputimage_height = (input_image_height - W_width_height + 1);
  this->Outputimage_channel = W_channel;
  this->Output_height = minibatch;
  this->Output_width =
      Outputimage_channel * Outputimage_height * Outputimage_width;
  this->Unroll_X_width = Outputimage_width * Outputimage_height;
  this->Unroll_X_height = W_width_height * W_width_height;
  this->X.resize(minibatch * input_image_height * input_image_width, 0);
  this->X_c.resize(minibatch * input_image_height * input_image_width, 0);
  this->Unroll_X.resize(W_width_height * W_width_height * Outputimage_width *
                            Outputimage_height,
                        0);
  this->Unroll_XT.resize(Outputimage_width * Outputimage_height *
                             W_width_height * W_width_height,
                         0);
  this->Unroll_X_c.resize(W_width_height * W_width_height * Outputimage_width *
                              Outputimage_height,
                          0);
  this->W_c.resize(W_channel * W_width_height * W_width_height, 0.5);
  this->W.resize(Outputimage_channel * W_width_height * W_width_height, 0.5);
  this->WT.resize(W_width_height * W_width_height * Outputimage_channel, 0.5);
  for (int i = 0; i < W_channel * W_width_height * W_width_height; i++) {
    W_c[i] = distribution(generator);
  }
  for (int i = 0; i < W_channel * W_width_height * W_width_height; i++) {
    W[i] = distribution(generator);
  }
  this->Output_c.resize(minibatch * Outputimage_channel * Outputimage_width *
                            Outputimage_height,
                        0);
  this->Output.resize(minibatch * Outputimage_channel * Outputimage_width *
                          Outputimage_height,
                      0);
  this->Wgrad_c.resize(Outputimage_channel * W_width_height * W_width_height,
                       0);
  this->Wgrad.resize(Outputimage_channel * W_width_height * W_width_height, 0);
  this->WgradTmp.resize(Outputimage_channel * W_width_height * W_width_height,
                        0);
}

void Convolution::forward_gpu() {
  dim3 threadsPerBlock(TILE_WIDTH, TILE_WIDTH);
  int bz = ceil((float)Outputimage_width / TILE_WIDTH) *
           ceil((float)Outputimage_height / TILE_WIDTH);
  if (bz == 0)
    bz = 1;
  dim3 numBlocks(minibatch, Outputimage_channel, bz);

  float *input_pointer = thrust::raw_pointer_cast(X.data());
  float *W_pointer = thrust::raw_pointer_cast(W.data());
  float *Output_pointer = thrust::raw_pointer_cast(Output.data());
  conv_layer_forward_gpu<<<numBlocks, threadsPerBlock>>>(
      input_pointer, W_pointer, Output_pointer, input_image_height,
      input_image_width, Outputimage_width, W_width_height,
      Outputimage_channel);
}

void Convolution::backward_gpu() {
  float *Output_pointer = thrust::raw_pointer_cast(Output.data());
  float *X_pointer = thrust::raw_pointer_cast(X.data());
  float *Wgrad_pointer = thrust::raw_pointer_cast(Wgrad.data());
  float *WgradTmp_pointer = thrust::raw_pointer_cast(WgradTmp.data());
  float *W_pointer = thrust::raw_pointer_cast(W.data());
  float *WT_pointer = thrust::raw_pointer_cast(WT.data());
  float *Unroll_X_pointer = thrust::raw_pointer_cast(Unroll_X.data());
  float *Unroll_XT_pointer = thrust::raw_pointer_cast(Unroll_XT.data());
  dim3 threadsPerBlock(TILE_WIDTH, TILE_WIDTH);
  dim3 numBlocks(
      ceil((float)Outputimage_width * Outputimage_height / TILE_WIDTH),
      ceil((float)Outputimage_channel /
           TILE_WIDTH)); // bx = O_WIDTH, by = O_HEIGHT
  dim3 numBlocks_back_dE_dW(ceil((float)Unroll_X_height / TILE_WIDTH),
                            ceil((float)Outputimage_channel /
                                 TILE_WIDTH)); // bx = O_WIDTH, by = O_HEIGH
  dim3 numBlocks_back_dE_dX(
      ceil((float)Unroll_X_width / TILE_WIDTH),
      ceil((float)Unroll_X_height / TILE_WIDTH)); // bx = O_WIDTH, by = O_HEIGH
  int num_threads = Outputimage_height * Outputimage_width;
  int num_blocks = ceil((float)num_threads / 1024);

  for (int i = 0; i < minibatch; i++) {
    // conv Wgrad
    // im2col

    unroll_kernel<<<num_blocks, 1024>>>(input_image_height, input_image_width,
                                        W_width_height, X_pointer,
                                        Unroll_X_pointer);

    // dL/dY * Unroll_X^t  = dY/dW
    transposeMatrix(Unroll_XT, Unroll_X, Unroll_X_height, Unroll_X_width);
    gemm_h<<<numBlocks_back_dE_dW, threadsPerBlock>>>(
        Output_pointer, Unroll_XT_pointer, WgradTmp_pointer,
        Outputimage_channel, Outputimage_height * Outputimage_width,
        Unroll_X_height, Outputimage_channel, Unroll_X_height);

    // sum Wgrad
    thrust::transform(Wgrad.begin(), Wgrad.end(), WgradTmp.begin(),
                      Wgrad.begin(), thrust::plus<float>());

    Output_pointer = Output_pointer + (Outputimage_channel *
                                       Outputimage_height * Outputimage_width);
    X_pointer = X_pointer + (input_image_height * input_image_width);
  }

  // divide by MINIBATCH
  thrust::transform(Wgrad.begin(), Wgrad.end(), Wgrad.begin(), div_h());

  //// gradient descent
  // bx*tx = idata_width*idata*height
  int blockDim = ceil((float)Outputimage_channel * Unroll_X_height / 1024);
  grad_descent<<<blockDim, 1024>>>(W_pointer, Wgrad_pointer,
                                   Outputimage_channel * Unroll_X_height);
}

// We will use 2D thread blocks
// Each thread block computing a tile of elements in output feature map
// Tile is defined as TILE_WIDTH * TILE_WIDTH
// A total of 256 threads per block for TILE_WIDTH =16
// Blocks will be organized into 3D grid
// Grid.X : N samples in the batch
// Grid.Y : M output channel of feature maps
// Grid.Z : location of the output tile inside output feature map
//• depend on the number of tiles in the horizontal and vertical dim

//// number of horizontal tiles per output map
// int W_grid = W_out / TILE_WIDTH;
// number of vertical tiles per output map
// int H_grid = H_out / TILE_WIDTH;

__global__ void conv_layer_forward_gpu(float *X, float *W, float *Y, int H_in,
                                       int W_in, int W_out, int K, int M) {
  int H_out = H_in - K + 1;
  int n, m, h, w, p, q;
  int W_grid = ceilf((float)W_out / TILE_WIDTH);
  if (W_grid == 0)
    W_grid = 1;
  n = blockIdx.x;
  m = blockIdx.y;
  h = (blockIdx.z / W_grid) * TILE_WIDTH + threadIdx.y;
  w = (blockIdx.z % W_grid) * TILE_WIDTH + threadIdx.x;
  // h and w is not center point, it's upper left corner point of Input image
  float acc = 0;
  // loop over KxK filter
  for (p = 0; p < K; p++) {
    for (q = 0; q < K; q++)
      if (h < H_out && w < W_out)
        acc = acc + X[n * (H_in * W_in) + (h + p) * (W_in) + (w + q)] *
                        W[m * (K * K) + p * (K) + q];
  }
  if (h < H_out && w < W_out) {
    Y[n * (M * H_out * W_out) + m * (H_out * W_out) + h * (W_out) + w] = acc;
  }
}

__global__ void unroll_kernel(int H_in, int W_in, int K, float *X,
                              float *X_unroll) {
  int s, h_out, w_out, h_unroll, w_unroll, h_base, p, q;
  int t = blockIdx.x * 1024 + threadIdx.x;
  int H_out = H_in - K + 1;
  int W_out = W_in - K + 1;
  int W_unroll = H_out * W_out;

  if (t < W_unroll) {
    s = t % W_unroll;                 // output height * output width
    h_out = s / W_out;                // output height
    w_out = s % W_out;                // output width
    w_unroll = h_out * W_out + w_out; // in conv1, max 28*28(s)
    for (p = 0; p < K; p++)
      for (q = 0; q < K; q++) {
        h_unroll = p * K + q;
        if ((h_out + p) < H_in && (w_out + q) < W_in)
          X_unroll[h_unroll * (W_unroll) + w_unroll] =
              X[(h_out + p) * W_in + w_out + q];
      }
  }
}
